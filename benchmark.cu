
#include <hip/hip_runtime.h>
#define SIZE (256 * (1 << 20))  // 256 MB
#define NUM_RUNS 10

std::pair<float, float> measureBandwidth() {
    void* hostData;
    void* deviceData;

    hostData = malloc(SIZE);
    hipMalloc(&deviceData, SIZE);

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    float totalHostToDeviceTime = 0.0f;
    float totalDeviceToHostTime = 0.0f;

    for (int run = 0; run < NUM_RUNS; ++run) {
        hipEventRecord(startEvent, 0);
        hipMemcpy(deviceData, hostData, SIZE, hipMemcpyHostToDevice);
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);

        float hostToDeviceTime;
        hipEventElapsedTime(&hostToDeviceTime, startEvent, stopEvent);
        totalHostToDeviceTime += hostToDeviceTime;

        hipEventRecord(startEvent, 0);
        hipMemcpy(hostData, deviceData, SIZE, hipMemcpyDeviceToHost);
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);

        float deviceToHostTime;
        hipEventElapsedTime(&deviceToHostTime, startEvent, stopEvent);
        totalDeviceToHostTime += deviceToHostTime;
    }

    float avgHostToDeviceBandwidth = (1000 * (float)SIZE * (float)NUM_RUNS) / (totalHostToDeviceTime * (float)(1 << 20));
    float avgDeviceToHostBandwidth = (1000 * (float)SIZE * (float)NUM_RUNS) / (totalDeviceToHostTime * (float)(1 << 20));

    free(hostData);
    hipFree(deviceData);

    return std::make_pair(avgHostToDeviceBandwidth, avgDeviceToHostBandwidth);
}