#include <iostream>
#include <hip/hip_runtime.h>

#include "benchmark.h"

int main() {
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);

    int runtimeVersion = 0;
    int driverVersion = 0;
    hipRuntimeGetVersion(&runtimeVersion);
    hipDriverGetVersion(&driverVersion);
    std::cout << "CUDA Runtime Version: " << runtimeVersion << std::endl;
    std::cout << "CUDA Driver Version: " << driverVersion << std::endl;

    if (error != hipSuccess) {
        std::cout << "Failed to get device count: " << hipGetErrorString(error) << std::endl;
        return 1;
    }

    if (deviceCount == 0) {
        std::cout << "\nNo NVIDIA GPU found." << std::endl;
    } else {
        std::cout << "\nFound " << deviceCount << " NVIDIA GPU(s)." << std::endl;

        // Print information about each GPU
        for (int i = 0; i < deviceCount; ++i) {
            hipDeviceProp_t deviceProp;
            error = hipGetDeviceProperties(&deviceProp, i);

            if (error == hipSuccess) {
                std::cout << "\nGPU " << i << " Information:" << std::endl;
                std::cout << "Name: " << deviceProp.name << std::endl;
                std::cout << "Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
                std::cout << "Total Global Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
                std::cout << "Multiprocessor Count: " << deviceProp.multiProcessorCount << std::endl;
            } else {
                std::cout << "Failed to get device properties for GPU " << i << ": " << hipGetErrorString(error) << std::endl;
            }
        }
    }

    std::pair<float, float> bandwidthValues = measureBandwidth();
    float hostToDeviceBandwidth = bandwidthValues.first;
    float deviceToHostBandwidth = bandwidthValues.second;

    std::cout << "\nHost to Device Bandwidth: " << hostToDeviceBandwidth << " MiB/s" << std::endl;
    std::cout << "Device to Host Bandwidth: " << deviceToHostBandwidth << " MiB/s" << std::endl;

    return 0;
}
